#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_kernels.h"

// 核函数实现
__global__ void compute_pq_distances(const uint8_t* d_vectors, const float* d_pq_dists, const uint32_t* d_ids,
                                     uint64_t n_ids, uint64_t n_chunks, float* d_dists_out) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx >= n_ids) return;

    // uint32_t id = d_ids[idx];
    // float dist = 0.0f;

    // // 通过ID索引对应的PQ向量
    // const uint8_t* pq_vector = d_vectors + id * n_chunks;
    // for (uint64_t chunk = 0; chunk < n_chunks; chunk++) {
    //     uint8_t pq_code = pq_vector[chunk];
    //     dist += d_pq_dists[chunk * 256 + pq_code];
    // }
    // d_dists_out[idx] = dist;

    if (idx < n_ids * n_chunks)
    {
        int id_offset = idx / n_chunks;
        int ch_offset = idx % n_chunks;
        float *ptr = (float *)d_pq_dists;
        ptr += 256 * ch_offset;
        uint32_t v_id = d_ids[id_offset];
        uint32_t offset = sizeof(uint8_t) * v_id * n_chunks + ch_offset;
        // uint32_t offset = v_id * n_chunks + ch_offset;
        atomicAdd(&d_dists_out[id_offset], ptr[d_vectors[offset]]);
    }
}

void call_compute_pq_distances(const uint8_t* d_vectors, const float* d_pq_dists, const uint32_t* d_ids,
                               uint64_t n_ids, uint64_t n_chunks, float* d_dists_out) {
    // dim3 threadsPerBlock(256);
    // dim3 numBlocks((n_ids * n_chunks + threadsPerBlock.x - 1) / threadsPerBlock.x);

    int block = 256;
    int grid = (n_ids * n_chunks + block - 1) / block;
    compute_pq_distances<<<grid, block>>>(
        d_vectors, d_pq_dists, d_ids, n_ids, n_chunks, d_dists_out);
    hipDeviceSynchronize();
}